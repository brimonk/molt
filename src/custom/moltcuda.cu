#include "hip/hip_runtime.h"
/*
 * Brian Chrzanowski
 * Tue Nov 19, 2019 21:02
 *
 * NOTE (brian)
 *
 * - The threaded implementation can rely on memory where the CUDA model
 *   can't _really_. IE, the working storage allocated by "molt.h" won't
 *   do our graphics card any good. Because the library (even the
 *   custom routine) expects that it will have the result of a given operation,
 *   reorg or step in the destination hunk of memory, we'll have to copy it
 *   to and from the graphics card.
 *
 *   There are some optimizations that we can do though.
 *
 *   1) We don't ever expect the values for the v and w weights to ever change.
 *      So, we can allocate storage for those at the beginning of the journey,
 *      and keep those allocated until the library calls "molt_custom_close".
 *
 *   2) Unfortunately, the library expects it to be able to give us source
 *      and destination memory hunks as it needs to. Because of this,
 *      the custom implementation can't just keep the problem state on-device
 *      the entire time, as that would require an equivalent amount of
 *      allocations to the host device.
 *
 *      To get around this, on every custom library call (reorg and sweep), we
 *      can compare the work, dst, and src pointers to what we got "last" time.
 *      If they're different, we need to copy from host to device, to update
 *      what the device is using to perform the operation.
 *
 * TODO (brian)
 * 1. Test with copying from host to device and host to device every time
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <float.h>
#include <limits.h>
#include <math.h>

#include "../common.h"
#define MOLT_IMPLEMENTATION
#include "../molt.h"

struct molt_custommod_t {
	f64 *d_src, *d_work, *d_dst;
	f64 *l_src, *l_dst; // the last pointers we've seen

	struct molt_cfg_t config;

	f64 *d_v[6];
	f64 *d_w[6];
	f64 *h_v[6];
	f64 *h_w[6];
};

static struct molt_custommod_t g_mod;

/* alloc_and_copy : allocs space on device, copies 'size' bytes from host */
int alloc_and_copy(f64 **d, f64 **newh, f64 *oldh, size_t size);
/* copy_if_needed : copies from host to device if needed, returns device ptr */
f64 *copy_if_needed(f64 *device, f64 *last_host, f64 *curr_host, size_t size);

/* cuda_gfquad_m : green's function quadriture on the input vector (CUDA) */
__device__ void cuda_gfquad_m(f64 *dst, f64 *src, f64 dnu, f64 *wl, f64 *wr, s64 len, s32 M);

/* molt_makel : applies dirichlet boundary conditions to the line in (CUDA) */
__device__ void cuda_makel(f64 *src, f64 *vl, f64 *vr, f64 minval, s64 len);

/* cuda_sweep : the cuda parallel'd sweeping function */
__global__ void cuda_sweep(f64 *dst, f64 *src, f64 *vl, f64 *vr, f64 *wl, f64 *wr, f64 minval, f64 dnu, s32 M, ivec3_t dim);

/* mk_genericidx : retrieves a generic index from input dimensionality */
__device__ u64 cuda_genericidx(ivec3_t ival, ivec3_t idim, cvec3_t order);

/* cuda_reorg : the cuda parallel'd transposition function */
__global__ void cuda_reorg(f64 *dst, f64 *src, f64 *work, ivec3_t dim, cvec3_t src_ord, cvec3_t dst_ord);

/* alloc_and_copy : allocs space on device, copies 'size' bytes from host */
int alloc_and_copy(f64 **d, f64 **newh, f64 *oldh, size_t size)
{
	hipError_t err;

	err = hipMalloc((void **)d, size);
	if (err != hipSuccess) {
		return -1;
	}

	err = hipMemcpy((void *)*d, (void *)oldh, size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		return -1;
	}

	// save the host pointer for later use
	*newh = oldh;

	return 0;
}

/* copy_if_needed : if needed, updates host pointer and data */
f64 *copy_if_needed(f64 *device, f64 **last_host, f64 *curr_host, size_t size)
{
	if ((*last_host) != curr_host) {
		*last_host = curr_host;
		hipMemcpy(device, curr_host, size, hipMemcpyHostToDevice);
	}

	return device;
}

/* molt_custom_init : intializes the custom module */
extern "C" __declspec(dllexport)
int molt_custom_open(struct molt_custom_t *custom)
{
	u64 elements;
	struct molt_cfg_t *cfg;
	hipError_t err;
	ivec3_t pinc;
	ivec3_t points;
	int rc;

	memset(&g_mod, 0, sizeof(struct molt_custommod_t));

	// snag a copy of the config structure we'll use
	// thoughout the module's lifetime
	memcpy(&g_mod.config, custom->cfg, sizeof(struct molt_cfg_t));

	cfg = &g_mod.config;

	molt_cfg_parampull_xyz(cfg, pinc, MOLT_PARAM_PINC);
	molt_cfg_parampull_xyz(cfg, points, MOLT_PARAM_POINTS);
	elements = pinc[0] * (u64)pinc[1] * pinc[2];

	rc = alloc_and_copy(&g_mod.d_v[0], &g_mod.h_v[0], custom->vlx, points[0] * sizeof(f64));
	if (rc < 0) { return -1; }
	rc = alloc_and_copy(&g_mod.d_v[1], &g_mod.h_v[1], custom->vrx, points[0] * sizeof(f64));
	if (rc < 0) { return -1; }
	rc = alloc_and_copy(&g_mod.d_v[2], &g_mod.h_v[2], custom->vly, points[1] * sizeof(f64));
	if (rc < 0) { return -1; }
	rc = alloc_and_copy(&g_mod.d_v[3], &g_mod.h_v[3], custom->vry, points[1] * sizeof(f64));
	if (rc < 0) { return -1; }
	rc = alloc_and_copy(&g_mod.d_v[4], &g_mod.h_v[4], custom->vlz, points[2] * sizeof(f64));
	if (rc < 0) { return -1; }
	rc = alloc_and_copy(&g_mod.d_v[5], &g_mod.h_v[5], custom->vrz, points[2] * sizeof(f64));
	if (rc < 0) { return -1; }

	rc = alloc_and_copy(&g_mod.d_w[0], &g_mod.h_w[0], custom->wlx, cfg->x_params[MOLT_PARAM_POINTS] * (cfg->spaceacc + 1) * sizeof(f64));
	if (rc < 0) { return -1; }
	rc = alloc_and_copy(&g_mod.d_w[1], &g_mod.h_w[1], custom->wrx, cfg->x_params[MOLT_PARAM_POINTS] * (cfg->spaceacc + 1) * sizeof(f64));
	if (rc < 0) { return -1; }
	rc = alloc_and_copy(&g_mod.d_w[2], &g_mod.h_w[2], custom->wly, cfg->y_params[MOLT_PARAM_POINTS] * (cfg->spaceacc + 1) * sizeof(f64));
	if (rc < 0) { return -1; }
	rc = alloc_and_copy(&g_mod.d_w[3], &g_mod.h_w[3], custom->wry, cfg->y_params[MOLT_PARAM_POINTS] * (cfg->spaceacc + 1) * sizeof(f64));
	if (rc < 0) { return -1; }
	rc = alloc_and_copy(&g_mod.d_w[4], &g_mod.h_w[4], custom->wlz, cfg->z_params[MOLT_PARAM_POINTS] * (cfg->spaceacc + 1) * sizeof(f64));
	if (rc < 0) { return -1; }
	rc = alloc_and_copy(&g_mod.d_w[5], &g_mod.h_w[5], custom->wrz, cfg->z_params[MOLT_PARAM_POINTS] * (cfg->spaceacc + 1) * sizeof(f64));
	if (rc < 0) { return -1; }

	// because we don't need to copy from the host, we'll just use cuda funcs
	err = hipMalloc(&g_mod.d_src, elements * sizeof(f64));
	if (err != hipSuccess) { return -1; }
	err = hipMalloc(&g_mod.d_dst, elements * sizeof(f64));
	if (err != hipSuccess) { return -1; }
	err = hipMalloc(&g_mod.d_work, elements * sizeof(f64));
	if (err != hipSuccess) { return -1; }

	return 0;
}

/* molt_custom_close : cleans up the custom module */
extern "C" __declspec(dllexport)
int molt_custom_close(struct molt_custom_t *custom)
{
	int i;

	for (i = 0; i < 6; i++) {
		hipFree(g_mod.d_v[i]);
		hipFree(g_mod.d_w[i]);
	}

	hipFree(g_mod.d_src);
	hipFree(g_mod.d_dst);
	hipFree(g_mod.d_work);

	memset(&g_mod, 0, sizeof(g_mod));

	return 0;
}

/* cuda_vect_mul : perform element-wise vector multiplication */
__device__
f64 cuda_vect_mul(f64 *veca, f64 *vecb, s32 veclen)
{
	f64 val;
	s32 i;

	for (val = 0, i = 0; i < veclen; i++) {
		val += veca[i] * vecb[i];
	}

	return val;
}


/* cuda_gfquad_m : green's function quadriture on the input vector (CUDA) */
__device__ void cuda_gfquad_m(f64 *dst, f64 *src, f64 dnu, f64 *wl, f64 *wr, s64 len, s32 M)
{
	/* out and in's length is defined by hunklen */
	f64 IL, IR;
	s32 iL, iR, iC, M2, N;
	s32 i;

	IL = 0;
	IR = 0;
	M2 = M / 2;
	N = len - 1;

	M++;

	iL = 0;
	iC = -M2;
	iR = len - M;

	/* left sweep */
	for (i = 0; i < M2; i++) {
		IL = dnu * IL + cuda_vect_mul(&wl[i * M] , &src[iL], M);
		dst[i + 1] = dst[i + 1] + IL;
	}

	for (; i < N - M2; i++) {
		IL = dnu * IL + cuda_vect_mul(&wl[i * M], &src[i + 1 + iC], M);
		dst[i + 1] = dst[i + 1] + IL;
	}

	for (; i < N; i++) {
		IL = dnu * IL + cuda_vect_mul(&wl[i * M], &src[iR], M);
		dst[i + 1] = dst[i + 1] + IL;
	}

	/* right sweep */
	for (i = N - 1; i > N - 1 - M2; i--) {
		IR = dnu * IR + cuda_vect_mul(&wr[i * M], &src[iR], M);
		dst[i] = dst[i] + IR;
	}

	for (; i >= M2; i--) {
		IR = dnu * IR + cuda_vect_mul(&wr[i * M], &src[i + 1 + iC], M);
		dst[i] = dst[i] + IR;
	}

	for (; i >= 0; i--) {
		IR = dnu * IR + cuda_vect_mul(&wr[i * M], &src[iL], M);
		dst[i] = dst[i] + IR;
	}

	// I = I / 2
	for (i = 0; i < len; i++)
		dst[i] /= 2;
}

/* molt_makel : applies dirichlet boundary conditions to the line in (CUDA) */
__device__ void cuda_makel(f64 *src, f64 *vl, f64 *vr, f64 minval, s64 len)
{
	/*
	 * molt_makel applies dirichlet boundary conditions to the line in place
	 *
	 * Executes this:
	 * w = w + ((wa - w(1)) * (vL - dN * vR) + (wb - w(end)) * (vR - dN * vL))
	 *			/ (1 - dN ^ 2)
	 *
	 * NOTE(s)
	 * wa and wb are left here as const scalars for future expansion of boundary
	 * conditions.
	 */

	f64 wa_use, wb_use, wc_use;
	f64 val;
	s64 i;

	const f64 wa = 0;
	const f64 wb = 0;

	// * wa_use - w(1)
	// * wb_use - w(end)
	// * wc_use - 1 - dN ^ 2
	wa_use = wa - src[0];
	wb_use = wb - src[len - 1];
	wc_use = 1 - pow(minval, 2);

	for (i = 0; i < len; i++) {
		val  = wa_use * vl[i] - minval * vr[i];
		val += wb_use * vr[i] - minval * vl[i];
		val /= wc_use;
		src[i] += val;
	}
}

/* cuda_sweep : the cuda parallel'd sweeping function */
__global__ void cuda_sweep(f64 *dst, f64 *src, f64 *vl, f64 *vr, f64 *wl, f64 *wr, f64 minval, f64 dnu, s32 M, ivec3_t dim)
{
	/*
	 * NOTE (brian)
	 * This function, while idiomatic CUDA, might seem a bit weird. Because
	 * launching the kernel in higher dimensions is, honestly, difficult to
	 * think about, I chose to solve the slightly easier problem, that is,
	 * launch the kernel in "a single dimension", with "Y by Z (dim[1] * dim[2])
	 * threads, then use our single dimension launch parameters to determine
	 * how far we're into the volume, using the IDX3D macro.
	 */

	u64 y, z, i;

	// first, get our thread number (thread 0, thread 1, thread 500, etc)
	i = threadIdx.x + blockDim.x * blockIdx.x;

	// use our volume dimensionality to determine the REAL y and z values from that
	// this assumes that we think about the problem in a "2D" sense
	y = i % dim[1];
	z = i / dim[1];

	i = IDX3D(0, y, z, dim[1], dim[2]);

	// don't perform the computation if we're out of range
	if (((u64)dim[0] * dim[1] * dim[2]) < i) {
		return;
	}

	// now that we have this thread's starting point, perform the algorithm
	// on this thread, for this row in x
	cuda_gfquad_m(dst + i, src + i, dnu, wl, wr, dim[0], M);
	cuda_makel(dst + i, vl, vr, minval, dim[0]);
}

/* molt_custom_sweep : performs a threaded sweep across the mesh in the dimension specified */
extern "C" __declspec(dllexport)
void molt_custom_sweep(f64 *dst, f64 *src, f64 *work, ivec3_t dim, cvec3_t ord, pdvec6_t params, dvec3_t dnu, s32 M)
{
	/*
	 * NOTE (brian)
	 * Because of the interface being defined how it is, we first have to
	 * create our mapping from host pointers, to dst pointers.
	 *
	 * We don't care about what the work pointer is because the device has to
	 * have its own working memory anyways, AND we're just going to memset it
	 * to 0 to begin with anyways.
	 *
	 * TODO (brian)
	 * - it might be worth making these function return ints and checking for
	 *   errors. Ideally, these won't error, but I suppose you wouldn't
	 *   know until you checked..
	 */

	f64 *d_src, *d_work, *d_dst;
	f64 *d_vl, *d_vr, *d_wl, *d_wr;
	f64 *h_vl, *h_vr, *h_wl, *h_wr;
	f64 usednu, minval;
	u64 elements, i;
	size_t bytes;
	dim3 block, grid;

	elements = (u64)dim[0] * dim[1] * dim[2];
	bytes = elements * sizeof(f64);

	// copy the bytes from host to device if the pointers have changed
	// since last time
	d_src  = copy_if_needed(g_mod.d_src, &g_mod.l_src, src, bytes);
	d_dst  = copy_if_needed(g_mod.d_dst, &g_mod.l_dst, dst, bytes);
	d_work = g_mod.d_work;

	// find our v and w weights on the device
	h_vl = params[0];
	h_vr = params[1];
	h_wl = params[2];
	h_wr = params[3];
	d_vl = NULL;
	d_vr = NULL;
	d_wl = NULL;
	d_wr = NULL;

	for (i = 0; i < 6; i++) {
		if (!d_vl && g_mod.h_v[i] == h_vl) {
			d_vl = g_mod.d_v[i];
		}
		if (!d_vr && g_mod.h_v[i] == h_vr) {
			d_vr = g_mod.d_v[i];
		}
		if (!d_wl && g_mod.h_w[i] == h_wl) {
			d_wl = g_mod.d_w[i];
		}
		if (!d_wr && g_mod.h_w[i] == h_wr) {
			d_wr = g_mod.d_w[i];
		}
	}

	// find the minval (dN in Matlab)
	// NOTE (brian) this should have the same dimensionality all the time
	for (i = 0, minval = DBL_MAX; i < dim[0]; i++) {
		if (h_vl[i] < minval)
			minval = h_vl[i];
	}

	// determine the correct dnu to use
	switch(ord[0]) {
	case 'x':
		usednu = dnu[0];
		break;
	case 'y':
		usednu = dnu[1];
		break;
	case 'z':
		usednu = dnu[2];
		break;
	default: // assert here? it's an illegal parameter
		break;
	}

	hipMemset(d_work, 0, bytes);

	u64 threads, blocks, iterations;

	iterations = dim[1] * dim[2];
	threads = 256;
	blocks = (iterations + threads - 1) / threads;

	// Launch kernel with dimensionality Y by Z, to sweep through the volume in a plane.
	// init dimensionality dim3s, launch our kernel, then wait for the sync
	cuda_sweep<<<blocks, threads>>>(d_dst, d_src, d_vl, d_vr, d_wl, d_wr, minval, usednu, M, dim);
	hipDeviceSynchronize();

	// copy from device to host, so the library's expectations are met
	hipMemcpy((void *)dst, (void *)d_dst, bytes, hipMemcpyDeviceToHost);
}

/* cuda_genericidx : retrieves a generic index from input dimensionality */
__device__ u64 cuda_genericidx(ivec3_t ival, ivec3_t idim, cvec3_t order)
{
	/*
	 * NOTE (brian)
	 * This is just a copy, for the CUDA device, of the library function
	 * of a similar name.
	 */

	ivec3_t lval, ldim;
	s32 i;

	for (i = 0; i < 3; i++) {
		switch (order[i]) {
		case 'x':
			lval[i] = ival[0];
			ldim[i] = idim[0];
			break;
		case 'y':
			lval[i] = ival[1];
			ldim[i] = idim[1];
			break;
		case 'z':
			lval[i] = ival[2];
			ldim[i] = idim[2];
			break;
		}
	}

	return IDX3D(lval[0], lval[1], lval[2], ldim[1], ldim[2]);

}

/* cuda_reorg : the cuda parallel'd transposition function */
__global__ void cuda_reorg(f64 *dst, f64 *src, f64 *work, ivec3_t dim, cvec3_t src_ord, cvec3_t dst_ord)
{
	u64 src_i, dst_i;
	ivec3_t curr;

	curr[0] = threadIdx.x + blockDim.x * blockIdx.x;
	curr[1] = threadIdx.y + blockDim.y * blockIdx.y;
	curr[2] = threadIdx.z + blockDim.z * blockIdx.z;

	src_i = cuda_genericidx(curr, dim, src_ord);
	dst_i = cuda_genericidx(curr, dim, src_ord);

	dst[dst_i] = src[src_i];
}

/* molt_custom_reorg : reorganizes a 3d mesh from src to dst */
extern "C" __declspec(dllexport)
void molt_custom_reorg(f64 *dst, f64 *src, f64 *work, ivec3_t dim, cvec3_t src_ord, cvec3_t dst_ord)
{
	f64 *d_src, *d_work, *d_dst;
	u64 elements;
	dim3 grid, block;
	size_t bytes;

	elements = (u64)dim[0] * dim[1] * dim[2];
	bytes = elements * sizeof(f64);

	d_src = copy_if_needed(g_mod.d_src, &g_mod.l_src, src, bytes);
	// d_dst = copy_if_needed(g_mod.d_dst, &g_mod.l_src, src, bytes);
	d_dst  = g_mod.d_src;
	d_work = g_mod.d_work;

	hipMemset(d_work, 0, bytes);

	// Unlike the cuda sweep, where we launch the kernel in a
	// "single dimension", we launch this one 3 dimensions.
	block = dim3(64, 64, 64);
	grid = dim3(ceil(dim[0] / block.x), ceil(dim[1] / block.y), ceil(dim[2] / block.z));

	cuda_reorg<<<grid, block>>>(d_dst, d_src, d_work, dim, src_ord, dst_ord);

	hipDeviceSynchronize();

	hipMemcpy((void *)dst, (void *)d_dst, bytes, hipMemcpyDeviceToHost);
}

